#include <iostream>
#include <cstdio>
#include <memory>
#include "hip/hip_runtime.h"
#define PR
bool fcc(float x, float y, float threshold = 1e-4) {
  return abs(x - y) < threshold;
}

bool allclose(float*x, float*y, int s) {
	for(int i=0;i<s;i++){
		float fx = x[i];
		float fy = y[i];
		bool fc = fcc(fx, fy);
		if(!fc) {
			return false;
		}
	}
	return true;
}

__global__ void dotg(float*dst, float*src1, float*src2, int xa, int ya, int za, int wa) {
  //xy * zw -> xz * yw
  int ywa = ya * wa;
  for(int xi = 0; xi < xa; xi++) {
    for(int yi = 0; yi < ya; yi++) {
      for(int zi = 0; zi < za; zi++) {
        for(int wi = 0; wi < wa; wi++) {
          int xzi = xi * za + zi;
          int ywi = yi * wa + wi;
          int ii = xzi * ywa + ywi;
          int xyi = xi * ya + yi;
          int zwi = zi * wa + wi;
          dst[ii] = src1[xyi] * src2[zwi];
        }
      }
    }
  }
}

void dotc(float*dst, float*src1, float*src2, int xa, int ya, int za, int wa) {
	//xy * zw -> xz * yw
	int ywa = ya * wa;
	for(int xi = 0; xi < xa; xi++) {
		for(int yi = 0; yi < ya; yi++) {
			for(int zi = 0; zi < za; zi++) {
				for(int wi = 0; wi < wa; wi++) {
					int xzi = xi * za + zi;
					int ywi = yi * wa + wi;
					int ii = xzi * ywa + ywi;
					int xyi = xi * ya + yi;
					int zwi = zi * wa + wi;
	 				dst[ii] = src1[xyi] * src2[zwi];
	 			}
			}
		}
	}
}

double checkpointc(int us = 1e3) {
	static clock_t timer = -1;
	clock_t newtime = clock();
	double ustime = us * ((newtime - timer) / (double)CLOCKS_PER_SEC);
	timer = newtime;
	return ustime;
}
	

int main(int argc, char* argv[]) {
	const int defdim = 39;
	int xa = argc > 1 ? std::atoi(argv[1]) : defdim;
  int ya = argc > 2 ? std::atoi(argv[2]) : defdim;
	int za = argc > 3 ? std::atoi(argv[3]) : defdim;
  int wa = argc > 4 ? std::atoi(argv[4]) : defdim;
	int aa = xa * ya * za * wa;
	float* src1 = (float*)malloc(sizeof(float) * xa * ya);
	float* src2 = (float*)malloc(sizeof(float) * za * wa);
	float* dstc = (float*)malloc(sizeof(float)* xa * ya * za * wa);
	float* dsrc1, *dsrc2, *dstd;
	hipMalloc((void**)&dsrc1, sizeof(float) * xa * ya);
  hipMalloc((void**)&dsrc2, sizeof(float) * za * wa);
	hipMalloc((void**)&dstd, sizeof(float) * aa);
  /*float* dsrc1 = (float*)malloc(sizeof(float) * xa * ya);
  float* dsrc2 = (float*)malloc(sizeof(float) * za * wa);
	float* dstd = (float*)malloc(sizeof(float)* xa * ya * za * wa);*/
	float* dstc2 = (float*)malloc(sizeof(float)* aa);
	std::cout<<"------SRC1 / SRC2--------"<<std::endl;
	for(int xi = 0; xi < xa; xi++) {
		for(int yi = 0; yi < ya; yi++) {
			src1[xi * ya + yi] = xi * ya + yi;
			std::cout<<src1[xi * ya + yi]<<" ";
		}
		std::cout<<std::endl;
	}
	std::cout<<"+++++THRES+++++++"<<std::endl;
	  std::cout<<"------SRC1 / SRC2--------"<<std::endl;
  for(int zi = 0; zi < za; zi++) {
    for(int wi = 0; wi < wa; wi++) {
			src2[zi * wa + wi] = zi * wa + wi;
      std::cout<<src2[zi * wa + wi]<<" ";
    }
		std::cout<<std::endl;
	}
	std::cout<<"SRC ->CPU -> DST"<<std::endl;
	checkpointc();
	dotc(dstc, src1 ,src2, xa, ya, za, wa);
	double cstime = checkpointc();
	std::cout<<"******CPU DST*********"<<std::endl;
	int xza = xa * za;
	int ywa = ya * wa;
#if 0
	for(int xzi = 0; xzi < xza; xzi++) {
		for(int ywi = 0; ywi < ywa; ywi++) {
			std::cout<<dstc[xzi * ywa + ywi]<<" ";
		}
		std::cout<<std::endl;
	}
#endif
	dim3 grids, blocks;
	grids.x = 1;
	grids.y = 1;
	grids.z = 1;
	blocks.x = 1;
	blocks.y = 1;
	blocks.z = 1;
	std::cout<<"SRC -> GPU ->DST"<<std::endl;
	checkpointc();
	hipMemcpy(dsrc1, src1, sizeof(float) * xa * ya, hipMemcpyHostToDevice);
	hipMemcpy(dsrc2, src2, sizeof(float) * za * wa, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	clock_t pretime = clock();
	dotg<<<grids, blocks>>>(dstd, dsrc1, dsrc2, xa, ya, za, wa);
	hipDeviceSynchronize();
	clock_t aftime = clock();
	double coretime = 1e3 * ((aftime - pretime) / (double)CLOCKS_PER_SEC);
	hipMemcpy(dstc2, dstd, sizeof(float) * aa, hipMemcpyDeviceToHost);
	double dstime = checkpointc();
	std::cout<<"******GPU DST*******"<<std::endl;
#if 0
  for(int xzi = 0; xzi < xza; xzi++) {
    for(int ywi = 0; ywi < ywa; ywi++) {
      std::cout<<dstc2[xzi * ywa + ywi]<<" ";
    }
    std::cout<<std::endl;
  }
#endif
	printf("DOTMUL: %d^%d * %d^%d\n", xa, ya, za, wa);
	std::cout<<"CSTIME: "<<cstime<<std::endl;
	std::cout<<"CPY-DET: "<<dstime - coretime<<std::endl;
	std::cout<<"CORETIME: "<<coretime<<std::endl;
	bool alc = allclose(dstc, dstc2, aa);
	std::cout<<"ALC: "<<alc<<std::endl;
}
