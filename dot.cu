#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <memory>
#include <assert.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#define PR
#define CEIL(x,y) ((x+y-1)/y)
#define SWAPMAX 20
#define STREAMAX 20
// WHILE BIG STREAMCNT WILL CAUSE ERR (MAY LESS than 20 OR SO)
// MAX SUPPORTTING 400 * 400
bool fcc(float x, float y, float threshold = 1e-4) {
  return abs(x - y) < threshold;
}

// helper function that wraps CUDA API calls, reports any error and exits
void chkCUDAErr(hipError_t error_id)                                                                                                                  
{
  if (error_id != hipSuccess){
    printf("CUDA ERROR :::%s\n", hipGetErrorString(error_id));
    exit(EXIT_FAILURE);
  }
}

__device__ 
float filter[2][2] = {{0.7, 0.4}, 
											{0.8, 0.9}};

__constant__ 
float cfilter[2][2] = {{0.7, 0.4},
											{0.8, 0.9}};

bool allclose(float*x, float*y, int s) {
	for(int i=0;i<s;i++){
		float fx = x[i];
		float fy = y[i];
		bool fc = fcc(fx, fy);
		if(!fc) {
			std::cout<<"ALC ERR for id "<<i<<std::endl;
			std::cout<<"FX VS FY "<<fx<<" "<<fy<<std::endl;
			return false;
		}
	}
	return true;
}

void dotc(float*dst, float*src1, float*src2, int xa, int ya, int za, int wa) {
	//xy * zw -> xz * yw
	int ywa = ya * wa;
	for(int xi = 0; xi < xa; xi++) {
		for(int yi = 0; yi < ya; yi++) {
			for(int zi = 0; zi < za; zi++) {
				for(int wi = 0; wi < wa; wi++) {
					int xzi = xi + zi * xa;
					int ywi = yi + wi * ya;
					int ii = xzi * ywa + ywi;
					int xyi = xi * ya + yi;
					int zwi = zi * wa + wi;
	 				dst[ii] = src1[xyi] * src2[zwi];
	 			}
			}
		}
	}
}

__global__ void dotgs(float*dst, float*src1, float*src2, int xa, int ya, int za, int wa) {
  // xy * zw -> xz * yw
	// blockIdx.x = x, blockIdx.y = y, threadIdx.x = z, threadIdx.y = w
  int ywa = ya * wa;
	int xi = blockIdx.x;
	int yi = blockIdx.y;
	int ze = CEIL(za, blockDim.x);
	int we = CEIL(wa, blockDim.y);
	int zbase = threadIdx.x * ze;
	int wbase = threadIdx.y * we;
	for(int zi = zbase; zi < zbase + ze; zi++ )	{
		for(int wi = wbase; wi < wbase + we; wi++) {
			if(zi < za) {
				if(wi < wa) {
          int xzi = xi + zi * xa;
          int ywi = yi + wi * ya;
          int ii = xzi * ywa + ywi;
          int xyi = xi * ya + yi;
          int zwi = zi * wa + wi;
          dst[ii] = src1[xyi] * src2[zwi];
				}
			}
		}
	}
}

__global__ void dotgxs(float*dst, float*src1, float*src2, int xa, int ya, int za, int wa, int zbias, int wbias) {
	// xy * zw -> xz * yw
  // blockIdx.x = x, blockIdx.y = y, threadIdx.x = z, threadIdx.y = w
	const int swapsize = SWAPMAX * SWAPMAX;
	__shared__ float swapzone[swapsize];
  int ywa = ya * wa;
  int xi = blockIdx.x;
  int yi = blockIdx.y;
	int zas = min(za, SWAPMAX);
	int was = min(wa, SWAPMAX);
  int ze = CEIL(zas, blockDim.x);
  int we = CEIL(was, blockDim.y);
  int zbase = threadIdx.x * ze + zbias;
  int wbase = threadIdx.y * we + wbias;
	// printf("HITTING PRE: zbase / ze / wbase / we: %d %d %d %d\n", zbase, ze, wbase, we);
  for(int zi = zbase; zi < zbase + ze; zi++ ) {
    for(int wi = wbase; wi < wbase + we; wi++) {
			// printf("TRYING zi / wi vs za / wa: %d / %d vs %d / %d BINGO ? %d\n", zi, wi, za, wa, (zi < za) && (wi < wa));
      if(zi < zbias + zas && zi < za) {
        if(wi < wbias + was && wi < wa) {
					// printf("BINGOING LAYER 2\n");
          int xyi = xi * ya + yi;
          int zwi = zi * wa + wi;
					// printf("LAYER 2 STATUS 2\n");
					int zwibias = (zi - zbias) * was + (wi - wbias);
					// printf("LAYER 2 STATUS 3\n");
					// printf("ZWIBIAS %d = (%d * %d)\n", zwibias, zi - zbias, wi - wbias);
					assert(zwibias < swapsize);
          swapzone[zwibias] = src1[xyi] * src2[zwi];
					// printf("LAYER 2 STATUS 4\n");
					// printf("HITTING ZWIBIAS: %d\n", zwibias);
        }
      }
    }
  }
	__syncthreads();
  for(int zi = zbase; zi < zbase + ze; zi++ ) {
    for(int wi = wbase; wi < wbase + we; wi++) {
      if(zi < zbias + zas && zi < za) {
        if(wi < wbias + was && wi < wa) {
          int xzi = xi + zi * xa;
          int ywi = yi + wi * ya;
          int ii = xzi * ywa + ywi;
          int zwi = zi * wa + wi;
					int zwibias = (zi - zbias) * was + (wi - wbias);
          dst[ii] = swapzone[zwibias];
        }
      }
    }
  }
}

double checkpointc(int us = 1e3) {
	static clock_t timer = -1;
	clock_t newtime = clock();
	double ustime = us * ((newtime - timer) / (double)CLOCKS_PER_SEC);
	timer = newtime;
	return ustime;
}
	

int main(int argc, char* argv[]) {
	const int defdim = SWAPMAX;
	int persq = argc > 1 ? std::min(std::atoi(argv[1]), 32) : 8;
	int xa = argc > 2 ? std::atoi(argv[2]) : defdim;
  int ya = argc > 3 ? std::atoi(argv[3]) : xa;
	int za = argc > 4 ? std::atoi(argv[4]) : ya;
  int wa = argc > 5 ? std::atoi(argv[5]) : za;
	int aa = xa * ya * za * wa;
	float* src1 = (float*)malloc(sizeof(float) * xa * ya);
	float* src2 = (float*)malloc(sizeof(float) * za * wa);
	float* hsrc1, * hsrc2;
	hipError_t  AllocErr;
	AllocErr = hipHostMalloc((void**)&hsrc1, sizeof(float) * xa * ya);
	if(AllocErr == hipErrorOutOfMemory) {
		assert(false);
	}
  AllocErr = hipHostMalloc((void**)&hsrc2, sizeof(float) * za * wa);
  if(AllocErr == hipErrorOutOfMemory) {
    assert(false);
  }
	float* dstc = (float*)malloc(sizeof(float)* xa * ya * za * wa);
	float* dsrc1, *dsrc2, *dstd, *dstd2, *dstd3;
	hipMalloc((void**)&dsrc1, sizeof(float) * xa * ya);
  hipMalloc((void**)&dsrc2, sizeof(float) * za * wa);
	hipMalloc((void**)&dstd, sizeof(float) * aa);
	hipMalloc((void**)&dstd2, sizeof(float) * aa);
	hipMalloc((void**)&dstd3, sizeof(float) * aa);
  /*float* dsrc1 = (float*)malloc(sizeof(float) * xa * ya);
  float* dsrc2 = (float*)malloc(sizeof(float) * za * wa);
	float* dstd = (float*)malloc(sizeof(float)* xa * ya * za * wa);*/
	float* dstc2 = (float*)malloc(sizeof(float)* aa);
	float* dstc3 = (float*)malloc(sizeof(float)* aa);
	float* dstc4 = (float*)malloc(sizeof(float)* aa);
	std::cout<<"------SRC1 / SRC2--------"<<std::endl;
#if 0
	for(int xi = 0; xi < xa; xi++) {
		for(int yi = 0; yi < ya; yi++) {
			hsrc1[xi * ya + yi] = xi * ya + yi;
			src1[xi * ya + yi] = xi * ya + yi;
			std::cout<<src1[xi * ya + yi]<<" ";
		}
		std::cout<<std::endl;
	}
	std::cout<<"+++++THRES+++++++"<<std::endl;
	  std::cout<<"------SRC2--------"<<std::endl;
  for(int zi = 0; zi < za; zi++) {
    for(int wi = 0; wi < wa; wi++) {
			hsrc2[zi * wa + wi] = zi * wa + wi;
			src2[zi * wa + wi] = zi * wa + wi;
      std::cout<<src2[zi * wa + wi]<<" ";
    }
		std::cout<<std::endl;
	}
#endif
	std::cout<<"SRC ->CPU -> DST"<<std::endl;
	checkpointc();
	dotc(dstc, src1 ,src2, xa, ya, za, wa);
	double cstime = checkpointc();
	std::cout<<"******CPU DST*********"<<std::endl;
	int xza = xa * za;
	int ywa = ya * wa;
#if 0
	for(int xzi = 0; xzi < xza; xzi++) {
		for(int ywi = 0; ywi < ywa; ywi++) {
			std::cout<<dstc[xzi * ywa + ywi]<<" ";
		}
		std::cout<<std::endl;
	}
#endif
	dim3 grids, blocks;
	grids.x = xa;
	grids.y = ya;
	grids.z = 1;
	blocks.x = persq;
	blocks.y = persq;
	blocks.z = 1;
	std::cout<<"SRC -> GPU ->DST"<<std::endl;
	checkpointc();
	hipMemcpy(dsrc1, src1, sizeof(float) * xa * ya, hipMemcpyHostToDevice);
	hipMemcpy(dsrc2, src2, sizeof(float) * za * wa, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	clock_t pretime = clock();
	dotgs<<<grids, blocks>>>(dstd, dsrc1, dsrc2, xa, ya, za, wa);
	hipDeviceSynchronize();
	clock_t aftime = clock();
	double coretime = 1e3 * ((aftime - pretime) / (double)CLOCKS_PER_SEC);
	hipMemcpy(dstc2, dstd, sizeof(float) * aa, hipMemcpyDeviceToHost);
	double dstime = checkpointc();
	std::cout<<"******GPU DST*******"<<std::endl;
#if 0
  for(int xzi = 0; xzi < xza; xzi++) {
    for(int ywi = 0; ywi < ywa; ywi++) {
      std::cout<<dstc2[xzi * ywa + ywi]<<" ";
    }
    std::cout<<std::endl;
  }
#endif
	bool alc = allclose(dstc, dstc2, aa);
	printf("DOTMUL: %d^%d * %d^%d\n", xa, ya, za, wa);
	std::cout<<"CSTIME: "<<cstime<<std::endl;
	std::cout<<"CPY-DET: "<<dstime - coretime<<std::endl;
	std::cout<<"CORETIME: "<<coretime<<std::endl;
	std::cout<<"PARA GPU 0.05X"<<std::endl;
	std::cout<<"TOTAL SPLIT 12X"<<std::endl;
	std::cout<<"GPU BENCHMARK "<<cstime / coretime<<" X"<<std::endl;
	std::cout<<"GPU CPY BOOST "<<cstime / dstime<<" X"<<std::endl;
	std::cout<<"ALC: "<<alc<<std::endl;
	std::cout<<"XS NOW"<<std::endl;
	hipDeviceSynchronize();
	checkpointc();
	double xstime = 0;
	for(int zi = 0; zi < za; zi +=SWAPMAX) {
		for(int wi = 0; wi < wa; wi +=SWAPMAX) {
			std::cout<<"DUP CAL for zbias / wbias "<<zi<<" / "<<wi<<std::endl;
			dotgxs<<<grids, blocks>>>(dstd2, dsrc1, dsrc2, xa, ya, za, wa, zi, wi);
			// hipDeviceSynchronize();
		}
	}
	hipDeviceSynchronize();
  double sxstime = checkpointc();
  xstime += sxstime;
	hipMemcpy(dstc3, dstd2, sizeof(float) * aa, hipMemcpyDeviceToHost);
#if 0
  for(int xzi = 0; xzi < xza; xzi++) {
    for(int ywi = 0; ywi < ywa; ywi++) {
      std::cout<<dstc3[xzi * ywa + ywi]<<" ";
    }
    std::cout<<std::endl;
  }
#endif
	bool alc2 = allclose(dstc, dstc3, aa);
	std::cout<<"XS CORETIME "<< xstime<<std::endl;
	std::cout<<"XS BENCHMARK "<< cstime / xstime <<" X"<<std::endl;
	std::cout<<"GPU CPY BOOST "<<cstime / (xstime + (dstime - coretime))<<" X"<<std::endl;
	std::cout<<"XS ALC "<< alc2<<std::endl;
  hipStream_t stream[STREAMAX * STREAMAX];
	int streamido = (wa + SWAPMAX - 1) / SWAPMAX;
	for(int zi = 0; zi < za; zi += SWAPMAX) {
		for(int wi = 0; wi < wa; wi += SWAPMAX) {
			int zis = zi / SWAPMAX;
			int wis = wi / SWAPMAX;
			int streamid = zis * streamido + wis;
			chkCUDAErr(hipStreamCreate(&stream[streamid]));
		}
	}
	// 待优化
	checkpointc();
	hipMemcpy(dsrc1, hsrc1, sizeof(float) * xa * ya, hipMemcpyHostToDevice);
	hipMemcpy(dsrc2, hsrc2, sizeof(float) * xa * ya, hipMemcpyHostToDevice);
	float singlecpy = checkpointc();
	std::cout<<"SINGLE CPY "<<singlecpy<<std::endl;
  for(int zi = 0; zi < za; zi += SWAPMAX) {
    for(int wi = 0; wi < wa; wi += SWAPMAX) {
      int zis = zi / SWAPMAX;
      int wis = wi / SWAPMAX;
      int streamid = zis * streamido + wis;
			int zitop = std::min(zi + SWAPMAX, za);
			int wisize = std::min(SWAPMAX, wa - wi);
			int zisize = zitop - zi;
			/*for(int zii = zi; zii < zitop; zii++) {
				int cpybias = (zii * wa + wi) * sizeof(float);
				hipMemcpyAsync(dsrc2 + cpybias, hsrc2 + cpybias, sizeof(float) * wisize, hipMemcpyHostToDevice, stream[streamid]);
			}*/
			dotgxs<<<grids, blocks, 0, stream[streamid]>>>(dstd3, dsrc1 ,dsrc2, xa, ya, za, wa, zi, wi);
			int zimax = zi * xa + zisize * xa;
			int cpysize = wisize * ya;
			int ywi = wi * ya;
			// std::cout<<std::endl<<"STOPPING POWER for z/w "<<zi<<" "<<wi<<std::endl<<std::endl;
			for(int xzi = zi * xa; xzi < zimax; xzi++) {
				int cpybias = (xzi * ywa + ywi);
				// std::cout<<"WRITING BACK at "<<(xzi * ywa + ywi)<<" for "<<cpysize<<std::endl;
				hipMemcpyAsync(dstc4 + cpybias, dstd3 + cpybias, sizeof(float) * cpysize, hipMemcpyDeviceToHost, stream[streamid]);
			}
    }
  }
	hipMemcpyAsync(dstc4, dstd3, sizeof(float) * aa, hipMemcpyDeviceToHost, stream[0]);
	hipDeviceSynchronize();
	float halfrun = checkpointc();
	std::cout<<"HALF RUN "<<halfrun<<std::endl;
	// hipMemcpyAsync(dstc4,dstd3, sizeof(float) * aa, hipMemcpyDeviceToHost);
	float finalcpy = checkpointc();
	std::cout<<"FINAL CPY "<<finalcpy<<std::endl;
#if 0
  for(int xzi = 0; xzi < xza; xzi++) {
    for(int ywi = 0; ywi < ywa; ywi++) {
      std::cout<<dstc4[xzi * ywa + ywi]<<" ";
    }
    std::cout<<std::endl;
  }
#endif
	bool alc3 = allclose(dstc, dstc4, aa);
	std::cout<<"STEAM ALC "<<alc3<<std::endl;
	std::cout<<"STREAM BOOST "<<cstime / (halfrun + finalcpy + singlecpy) <<" X"<<std::endl;
  for(int zi = 0; zi < za; zi += SWAPMAX) {
    for(int wi = 0; wi < wa; wi += SWAPMAX) {
      int zis = zi / SWAPMAX;
      int wis = wi / SWAPMAX;
      int streamid = zis * streamido + wis;
      chkCUDAErr(hipStreamDestroy(stream[streamid]));
    }
  }
	free(src1);
	free(src2);
	hipHostFree(hsrc1);
	hipHostFree(hsrc2);
	free(dstc);
	free(dstc2);
	free(dstc3);
	free(dstc4);
	hipFree(dsrc1);
	hipFree(dsrc2);
	hipFree(dstd);
	hipFree(dstd2);
	hipFree(dstd3);
	int count;
	hipDeviceProp_t prob;
	hipGetDeviceCount(&count);
	for(int i=0; i< count; i++) {
		hipGetDeviceProperties(&prob, i);
		std::cout<<std::endl<<"DEV PRO "<<std::endl;
		std::cout<<"NAME "<<prob.name<<std::endl;
		std::cout<<"COMPUTE COMPABILITY "<<prob.major<<" X "<<prob.minor<<std::endl;
		std::cout<<"CLOCK RATE "<<prob.clockRate<<std::endl;
	}
}
